#include "hip/hip_runtime.h"
#include "../../include/error.cuh"
#include "force.h"
#include "mic.h"

struct LJ {
    real cutoff2;
    real e24s6;
    real e48s12;
    real e4s6;
    real e4s12;
};

static void __global__ gpu_find_force(LJ lj, int N, int *g_NN, int *g_NL, Box box,
                                      real *g_x, real *g_y, real *g_z, real *g_fx, real *g_fy, real *g_fz, real *g_pe) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        real fx = 0.0;
        real fy = 0.0;
        real fz = 0.0;
        real potential = 0.0;
        int NN = g_NN[i];
        real x_i = g_x[i];
        real y_i = g_y[i];
        real z_i = g_z[i];
        for(int k = 0; k < NN; ++k) {
            int j = g_NL[i + N * k];
            real x_ij = g_x[j] - x_i;
            real y_ij = g_y[j] - y_i;
            real z_ij = g_z[j] - z_i;
            apply_mic(box, &x_ij, &y_ij, &z_ij);
            real r2 = x_ij*x_ij + y_ij*y_ij + z_ij*z_ij;
            if(r2 > lj.cutoff2) { continue; }

            real r2inv = 1.0 / r2;
            real r4inv = r2inv * r2inv;
            real r6inv = r2inv * r4inv;
            real r8inv = r4inv * r4inv;
            real r12inv = r4inv * r8inv;
            real r14inv = r6inv * r8inv;
            real f_ij = lj.e24s6 * r8inv - lj.e48s12 * r14inv;
            potential += lj.e4s12 * r12inv - lj.e4s6 * r6inv;
            fx += f_ij * x_ij;
            fy += f_ij * y_ij;
            fz += f_ij * z_ij;
        }
        g_fx[i] = fx;
        g_fy[i] = fy;
        g_fz[i] = fz;
        g_pe[i] = potential * 0.5;
    }
}