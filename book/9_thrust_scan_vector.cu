#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <stdio.h>

int main() {
    int N = 10;
    thrust::device_vector<int> x(N, 0);
    thrust::device_vector<int> y(N, 0);
    for (int i = 0; i < x.size(); ++i) {
        x[i] = i + 1;
    }
    thrust::inclusive_scan(x.begin(), x.end(), y.begin());
    for (int i = 0; i < y.size(); ++i) {
        printf("%d ", (int) y[i]);
    }
    printf("\n");
    return 0;
}
