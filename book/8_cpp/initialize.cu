#include "hip/hip_runtime.h"
#include "initialize.cuh"
#include <stdlib.h>
#include <math.h>

static void scale_velocity(int N, real T_0, Atom *atom) {
    real *m = atom->m;
    real *vx = atom->vx;
    real *vy = atom->vy;
    real *vz = atom->vz;
    real temperature = 0.0;
    for(int n = 0; n < N; ++n) {
        real v2 = vx[n]*vx[n] + vy[n]*vy[n] + vz[n]*vz[n];
        temperature += m[n] * v2;
    }
    temperature /= 3.0 * K_B * N;
    real scale_factor = sqrt(T_0 / temperature);
    for(int n = 0; n < N; ++n) {
        vx[n] *= scale_factor;
        vy[n] *= scale_factor;
        vz[n] *= scale_factor;
    }
}

void initialize_position(int nx, real ax, Atom *atom) {
    atom->box[0] = ax * nx;
    atom->box[1] = ax * nx;
    atom->box[2] = ax * nx;
    atom->box[3] = atom->box[0] * 0.5;
    atom->box[4] = atom->box[1] * 0.5;
    atom->box[5] = atom->box[2] * 0.5;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real x0[4] = {0.0, 0.0, 0.5, 0.5};
}
