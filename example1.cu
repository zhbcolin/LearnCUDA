
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void my_first_kernel() {

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    printf("Hello World from GPU!\n");
}

// thread --> block --> grid
// SM stream multi-processor
int main() {

    printf("Hello World from CPU!\n");

    int block_size = 3;
    int grid_size = 2;

    my_first_kernel<<<grid_size,block_size>>>();
    hipDeviceSynchronize();

    return 0;
}