
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void my_first_kernel() {

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;

    printf("Hello World from thread(thread index: (%d, %d), block index: (%d, %d))!\n", tidy, tidx, bidy, bidx);
}

// thread --> block --> grid
// SM stream multi-processor
int main() {

    printf("Hello World from CPU!\n");

    //int block_size = 3;
    dim3 block_size(3, 3);
    // t00, t01. t02
    // t10, t11, t12
    // t20, t21, t22
    dim3 grid_size(2, 2);
    //int grid_size = 2;
    // b00, b01
    // b10, b11

    my_first_kernel<<<grid_size,block_size>>>();
    hipDeviceSynchronize();

    return 0;
}