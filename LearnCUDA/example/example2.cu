// index data
// z[i] = x[i] + y[i]
// for loop
// thread z[i]
// memory allocation
// memory copy    gpu mem != cpu mem
// kernel func
// memory copy

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// x[] + y[] = z[]
__global__ void vecAdd(const double *x, const double *y, double *z, int count) {
    const int index = blockDim.x * blockIdx.x + threadIdx.x;
    // t00 t01 t02 t10 t11 t12 t20 t21 t22
    if(index < count) {
        z[index] = x[index] + y[index];
    }
}

void vecAdd_cpu(const double *x, const double *y, double *z, int count) {
    for(int i = 0; i < count; ++i) {
        z[i] = x[i] + y[i];
    }
}

int main() {

    const int N = 1000;
    const int M = sizeof(double) * N;

    // cpu mem alloc
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);
    double *result_cpu = (double*) malloc(M);

    for(int i = 0; i < N; ++i) {
        h_x[i] = 1;
        h_y[i] = 2;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void**) &d_x, M);
    hipMalloc((void**) &d_y, M);
    hipMalloc((void**) &d_z, M);

    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    vecAdd<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);

    vecAdd_cpu(h_x, h_y, result_cpu, N);

    bool error = false;

    for(int i = 0; i < N; ++i) {
        if(fabs(result_cpu[i] - h_z[i]) > (1.0e-10)) {
            error = true;
        }
    }

    printf("Result: %s\n", error ? "Errors" : "Pass");

    free(h_x);
    free(h_y);
    free(h_z);
    free(result_cpu);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}