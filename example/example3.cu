//#include <stdio.h>
//#include <math.h>
//
//#define BLOCK_SIZE 16
//// a[][] * b[][] = c[][]
////
//
////                         b00 b01 b02 b03
////                         b10 b11 b12 b13
////                         b20 b21 b22 b23
////                         b30 b31 b32 b33
////
//// a00 a01 a02 a03         c00 c01 c02 c03
//// a10 a11 a12 a13         c10 c11 c12 c13
//// a20 a21 a22 a23         c20 c21 c22 c23
//// a30 a31 a32 a33         c30 c31 c32 c33
////
//// c21 = a20 * b01 + a21 * b11 + a22 * b21 + a23 * b31
//// a00 a01 a02 a03 a10 a11 a12 a13 a20 a21 a22 a23 a30 a31 a32 a33
//// 0   1   2   3   4   5   6   7   8   9   10  11  12  13  14  15
//// b00 b01 b02 b03 b10 b11 b12 b13 b20 b21 b22 b23 b30 b31 b32 b33
////
//// index = y * size + x
//// step 0 -> 3
//// a_index = y * size + step;
//// b_index = step * size + x;
//
//__global__ void gpu_matrix_mult(int *a, int *b, int *c, const int size) {
//    int y = blockDim.y * blockIdx.y + threadIdx.y;
//    int x = blockDim.x * blockIdx.x + threadIdx.x;
//    if(x < size && y < size) {
//        int tmp = 0;
//        for(int step = 0; step < size; ++step) {
//            tmp += a[y*size + step] * b[step * size + x];
//        }
//        c[y * size + x] = tmp;
//    }
//}
//
//void cpu_matrix_mult(int *a, int *b, int *c, const int size) {
//    for(int y = 0; y < size; ++y) {
//        for(int x = 0; x < size; ++x) {
//            int tmp = 0;
//            for(int step = 0; step < size; ++step) {
//                tmp += a[y*size + step] * b[step * size + x];
//            }
//            c[y * size + x] = tmp;
//        }
//    }
//}
//
//int main() {
//
//    const int matrix_size = 1000;
//    const int memsize = sizeof(int) * matrix_size * matrix_size;
//
//    int *h_a, *h_b, *h_c, *h_cc;
//    cudaMallocHost((void**)&h_a, memsize);
//    cudaMallocHost((void**)&h_b, memsize);
//    cudaMallocHost((void**)&h_c, memsize);
//    cudaMallocHost((void**)&h_cc, memsize);
//
//    for(int y = 0; y < matrix_size; ++y) {
//        for(int x = 0; x < matrix_size; ++x) {
//            h_a[y * matrix_size + x] = rand() % 1024;
//            h_b[y * matrix_size + x] = rand() % 1024;
//        }
//    }
//
//    int *d_a, *d_b, *d_c;
//    cudaMalloc((void**) &d_a, memsize);
//    cudaMalloc((void**) &d_b, memsize);
//    cudaMalloc((void**) &d_c, memsize);
//
//    cudaMemcpy(d_a, h_a, memsize, cudaMemcpyHostToDevice);
//    cudaMemcpy(d_b, h_b, memsize, cudaMemcpyHostToDevice);
//
//    unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
//    unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
//
//    dim3 dimGrid(grid_cols, grid_rows);
//    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // 1.gpu warp 32 2. <= 1024
//
//    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);
//
//    cudaMemcpy(h_c, d_c, memsize, cudaMemcpyDeviceToHost);
//
//    cpu_matrix_mult(h_a, h_b, h_cc, matrix_size);
//
//    bool error = false;
//    for(int y = 0; y < matrix_size; ++y) {
//        for(int x = 0; x < matrix_size; ++x) {
//            if(fabs(h_cc[y * matrix_size + x] - h_c[y * matrix_size + x]) > (1.0e-10)) {
//                error = true;
//            }
//        }
//    }
//    printf("Result: %s\n", error ? "Errors" : "Pass");
//
//    cudaFreeHost(h_a);
//    cudaFreeHost(h_b);
//    cudaFreeHost(h_c);
//    cudaFreeHost(h_cc);
//    cudaFree(d_a);
//    cudaFree(d_b);
//    cudaFree(d_c);
//
//    return 0;
//}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// a[][] * b[][] = c[][]
//
//                         b00 b01 b02 b03
//                         b10 b11 b12 b13
//                         b20 b21 b22 b23
//                         b30 b31 b32 b33
//
// a00 a01 a02 a03         c00 c01 c02 c03
// a10 a11 a12 a13         c10 c11 c12 c13     block(1, 0) -> shared memory
// a20 a21 a22 a23         c20 c21 c22 c23     c20 c21
// a30 a31 a32 a33         c30 c31 c32 c33     c30 c31
//
//                              b00 b01->  sub_b_step_0
//                              b10 b11
//
//                              b20 b21->  sub_b_step_1
//                              b30 b31
// sub_a_step_0 sub_a_step_1    sub_c
// a20 a21      a22 a23         c20 c21
// a30 a31      a32 a33         c30 c31
//
// sub_c = sub_a_step_0 * sub_b_step_0 + sub_a_step_1 * sub_b_step_1;
//
// for(int step =0; step < N/block_size; step++ )
//      load sub_a_step to shared memory;
//      load sub_b_step to shared memory;
//      tmp += sub_a_step_on_sharedmemory * sub_b_step_on_sharedmemory;
// sub_c = tmp;
//
// cudaMalloc -> global memory
// data global memory -> shared memory
// threads shared memory -> register
// shared memory SM(stream multi-processor) same block same shared memory
//
// c21 = a20 * b01 + a21 * b11 + a22 * b21 + a23 * b31
// a00 a01 a02 a03 a10 a11 a12 a13 a20 a21 a22 a23 a30 a31 a32 a33
// 0   1   2   3   4   5   6   7   8   9   10  11  12  13  14  15
// b00 b01 b02 b03 b10 b11 b12 b13 b20 b21 b22 b23 b30 b31 b32 b33

#define M 1000
#define N 500
#define K 1000

__managed__ int a[M*N];
__managed__ int b[N*K];
__managed__ int c_gpu[M*K];
__managed__ int c_cpu[M*K];

#define BLOCK_SIZE 16

__global__ void gpu_matrix(int* a, int* b, int* c, int m, int n, int k)
{
    __shared__ int sub_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sub_b[BLOCK_SIZE][BLOCK_SIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int tmp =0;
    int idx;
    for(int step=0; step <= n/BLOCK_SIZE; step++)
    {
        int step_x = step * BLOCK_SIZE + threadIdx.x;
        int step_y = y;
        idx = step_y * n + step_x;
        if(step_x >= n || step_y >= m)
        {
            sub_a[threadIdx.y][threadIdx.x] =0;
        }
        else
        {
            sub_a[threadIdx.y][threadIdx.x] = a[idx];
        }

        step_x = x;
        step_y = step * BLOCK_SIZE + threadIdx.y;
        idx = step_y * k +step_x;
        if(step_x >= k || step_y >= n)
        {
            sub_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            sub_b[threadIdx.y][threadIdx.x] = b[idx];
        }

        __syncthreads();

        for(int i = 0; i < BLOCK_SIZE; i++)
        {
            tmp +=sub_a[threadIdx.y][i] * sub_b[i][threadIdx.x];
        }
        __syncthreads();
    }

    if ( x < k && y < m)
    {
        c[y*k + x] = tmp;
    }
}

void cpu_matrix(int* a, int* b, int* c, int m, int n, int k)
{
    for( int y = 0; y < m; y++)
    {
        for(int x = 0; x < k; x++)
        {
            int tmp = 0;
            for(int step =0; step < n; step++)
            {
                tmp += a[y*n + step] * b[step*k + x];
            }
            c[y * k + x] = tmp;
        }
    }
}

int main()
{
    for(int y=0; y<M; ++y)
    {
        for(int x=0; x<N; ++x)
        {
            a[y * N + x] = rand()%1024;
        }
    }

    for(int y=0; y<N; ++y)
    {
        for(int x=0; x<K; ++x)
        {
            b[y*K + x] = rand()%1024;
        }
    }

    unsigned int grid_x = (K + BLOCK_SIZE -1)/BLOCK_SIZE;
    unsigned int grid_y = (M + BLOCK_SIZE -1)/BLOCK_SIZE;

    dim3 dimGrid(grid_x, grid_y);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix<<<dimGrid, dimBlock>>>(a, b, c_gpu, M, N, K);

    cpu_matrix(a, b, c_cpu, M, N, K);

    bool errors = false;

    for(int y=0; y<M; y++)
    {
        for(int x=0; x<K; x++)
        {
            if(fabs(c_cpu[y*K + x] - c_gpu[y*K+x]) > (1.0e-10))
            {
                errors = true;
                printf("c_cpu: %d. c_gpu: %d", c_cpu[y*K + x], c_gpu[y*K+x]);
            }
        }
    }

    printf("Result: %s\n", errors?"Error":"Pass");

    return 0;
}