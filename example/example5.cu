#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "LearnCUDA/include/error.cuh"

#define BLOCK_SIZE 32
// a[][] * b[][] = c[][]
//

//                         b00 b01 b02 b03
//                         b10 b11 b12 b13
//                         b20 b21 b22 b23
//                         b30 b31 b32 b33
//
// a00 a01 a02 a03         c00 c01 c02 c03
// a10 a11 a12 a13         c10 c11 c12 c13
// a20 a21 a22 a23         c20 c21 c22 c23
// a30 a31 a32 a33         c30 c31 c32 c33
//
// c21 = a20 * b01 + a21 * b11 + a22 * b21 + a23 * b31
// a00 a01 a02 a03 a10 a11 a12 a13 a20 a21 a22 a23 a30 a31 a32 a33
// 0   1   2   3   4   5   6   7   8   9   10  11  12  13  14  15
// b00 b01 b02 b03 b10 b11 b12 b13 b20 b21 b22 b23 b30 b31 b32 b33
//
// index = y * size + x
// step 0 -> 3
// a_index = y * size + step;
// b_index = step * size + x;

__global__ void gpu_matrix_mult(int *a, int *b, int *c, const int size) {
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < size && y < size) {
        int tmp = 0;
        for(int step = 0; step < size; ++step) {
            tmp += a[y*size + step] * b[step * size + x];
        }
        c[y * size + x] = tmp;
    }
}

void cpu_matrix_mult(int *a, int *b, int *c, const int size) {
    for(int y = 0; y < size; ++y) {
        for(int x = 0; x < size; ++x) {
            int tmp = 0;
            for(int step = 0; step < size; ++step) {
                tmp += a[y*size + step] * b[step * size + x];
            }
            c[y * size + x] = tmp;
        }
    }
}

int main() {

    const int matrix_size = 1000;
    const int memsize = sizeof(int) * matrix_size * matrix_size;

    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void**)&h_a, memsize);
    hipHostMalloc((void**)&h_b, memsize);
    hipHostMalloc((void**)&h_c, memsize);
    hipHostMalloc((void**)&h_cc, memsize);

    for(int y = 0; y < matrix_size; ++y) {
        for(int x = 0; x < matrix_size; ++x) {
            h_a[y * matrix_size + x] = rand() % 1024;
            h_b[y * matrix_size + x] = rand() % 1024;
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, memsize);
    hipMalloc((void**) &d_b, memsize);
    hipMalloc((void**) &d_c, memsize);
    hipError_t error_code;

    hipEvent_t start, stop_cpu, stop_gpu;
    CHECK(hipEventCreate(&start));
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);

    CHECK(hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice));

    unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // 1.gpu warp 32 2. <= 1024

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);
    error_code = hipGetLastError();
    if(error_code != hipSuccess) {
        printf("Errors info: %s\n", hipGetErrorString(error_code));
        printf("FILE: %s\n", __FILE__);
        printf("LINE: %d\n", __LINE__);
    }

    hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_mult(h_a, h_b, h_cc, matrix_size);

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    printf("GPU time: %.2f ms\n", time_gpu);
    printf("CPU time: %.2f ms\n", time_cpu);

    hipEventDestroy(start);
    hipEventDestroy(stop_gpu);
    hipEventDestroy(stop_cpu);

    bool error = false;
    for(int y = 0; y < matrix_size; ++y) {
        for(int x = 0; x < matrix_size; ++x) {
            if(fabs(h_cc[y * matrix_size + x] - h_c[y * matrix_size + x]) > (1.0e-10)) {
                error = true;
            }
        }
    }
    printf("Result: %s\n", error ? "Errors" : "Pass");

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}