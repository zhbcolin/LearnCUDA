// matrix transpose
//                                      t56 t57 t58
// in       b00 b01 b02 | b03 b04 b05 | b06 b07 b08     B[6][9]
//          b10 b11 b12 | b13 b14 b15 | b16 b17 b18
//          b20 b21 b22 | b23 b24 b25 | b26 b27 b28
//          ------------+-------------+------------
//          b30 b31 b32 | b33 b34 b35 | b36 b37 b38
//          b40 b41 b42 | b43 b44 b45 | b46 b47 b48
//          b50 b51 b52 | b53 b54 b55 | b56 b57 b58     threadIdx.x = 1, threadIdx.y = 2;
//                                                      block 1, 2
//
// out      b00 b10 b20 | b30 b40 b50
//          b01 b11 b21 | b31 b41 b51
//          b02 b12 b22 | b32 b42 b52
//          ------------+------------
//          b03 b13 b23 | b33 b43 b53
//          b04 b14 b24 | b34 b44 b54
//          b05 b15 b25 | b35 b45 b55
//          ------------+------------
//          b06 b16 b26 | b36 b46 b56
//          b07 b17 b27 | b37 b47 b57
//          b08 b18 b28 | b38 b48 b58                   block 2, 1
// shared memory
// t57 read b57 from global memory to shared memory
// t57 read b48 from shared memory
// t57 write b48 to global memory


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 32
#define M 3000
#define N 1000

__managed__ int matrix[N][M];
__managed__ int gpu_result[M][N];
__managed__ int cpu_result[M][N];

__global__ void gpu_matrix_transpose(int in[N][M], int out[M][N]) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x < M && y < N) {
        out[x][y] = in[y][x];
    }
}

__global__ void gpu_shared_matrix_transpose(int in[N][M], int out[M][N]) {
   int y = threadIdx.y + blockDim.y * blockIdx.y;
   int x = threadIdx.x + blockDim.x * blockIdx.x;

   __shared__ int ken[BLOCK_SIZE+1][BLOCK_SIZE+1]; // ken[32] warp

   if(x < M && y < N) {
       ken[threadIdx.y][threadIdx.x] = in[y][x];
   }

   __syncthreads();

   int x1 = threadIdx.x + blockDim.y * blockIdx.y;
   int y1 = threadIdx.y + blockDim.x * blockIdx.x;
   if(x1 < N && y1 < M) {
       out[y1][x1] = ken[threadIdx.x][threadIdx.y]; // 32 bank
   }
}

void cpu_matrix_transpose(int in[N][M], int out[M][N]) {
    for(int y = 0; y < N; ++y) {
        for(int x = 0; x < M; ++x) {
            out[x][y] = in[y][x];
        }
    }
}

int main() {
    for(int y = 0; y < N; ++y) {
        for(int x = 0; x < M; ++x) {
            matrix[y][x] = rand() % 1024;
        }
    }

    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    dim3 dimGrid((M + BLOCK_SIZE - 1)/BLOCK_SIZE, (N + BLOCK_SIZE -1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    for(int i = 0; i < 20; ++i) {
        gpu_shared_matrix_transpose<<<dimGrid, dimBlock>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_transpose(matrix, cpu_result);

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool errors = false;
    for(int y = 0; y < M; ++y) {
        for(int x = 0; x < N; ++x) {
            if(fabs(cpu_result[y][x] - gpu_result[y][x]) > (1.0e-10)) {
                errors = true;
            }
        }
    }

    printf("Result: %s\n", errors ? "Error" : "Pass");
    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20.0);

    return 0;
}