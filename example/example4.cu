#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace std;
using namespace cv;

// Sobel:
// kernel  x00 x01 x02     m00 m01 m02 m03 m04    r00 r01 r02 r03 r04
//         x10 x11 x12     m10 m11 m12 m13 m14    r10 r11 r12 r13 r14
//         x20 x21 x22     m20 m21 m22 m23 m24    r20 r21 r22 r23 r24
//                         m30 m31 m32 m33 m34    r30 r31 r32 r33 r34
//                         m40 m41 m42 m43 m44    r40 r41 r42 r43 r44
//
// r11 = m00 * x00 + m01 * x01 + x02 * m02 + ... + x22 * m22
// Gx 1 0 -1     Gy 1  2  1
//    2 0 -2        0  0  0
//    1 0 -1       -1 -2 -1
// Gx_r11 = m00 + 2*m10 + m20 - m02 - 2*m12 - m22
// Gy_r11 = m00 + 2*m01 + m02 - m20 - 2*m21 - m22
// r11 = (Gx + Gy) / 2

__global__ void sobel_gpu(unsigned char* in, unsigned char* out, const int Height, const int Width) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int index = y * Width + x;

    int Gx = 0;
    int Gy = 0;
    unsigned char x0, x1, x2, x3, x4, x5, x6, x7, x8;
    // SM register


    if(x > 0 && x < (Width-1) && y > 0 && y < (Height-1)) {
        x0 = in[(y-1)*Width + (x-1)];
        x1 = in[(y-1)*Width + (x)];
        x2 = in[(y-1)*Width + (x+1)];
        x3 = in[(y)*Width + (x-1)];
        x4 = in[(y)*Width + (x)];;
        x5 = in[(y)*Width + (x+1)];
        x6 = in[(y+1)*Width + (x-1)];
        x7 =  in[(y+1)*Width + (x)];
        x8 =  in[(y+1)*Width + (x+1)];

        Gx = (x0 + 2*x3 + x6) - (x2 + 2*x5 + x7);
        Gy = (x0 + 2*x1 + x2) - (x6 + 2*x7 + x8);

        out[index] = (Gx + Gy) / 2;
    }
}

int main() {
    Mat img = imread("../res/lena.png", 0);
    int height = img.rows;
    int width = img.cols;

    Mat gaussImg;
    GaussianBlur(img, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    Mat dst_gpu(height, width, CV_8UC1, Scalar(0));

    int memsize = height * width * sizeof(unsigned char);

    unsigned char* in_gpu;
    unsigned char* out_gpu;

    hipMalloc((void**) &in_gpu, memsize);
    hipMalloc((void**) &out_gpu, memsize);

    dim3 threadsPerBlock(32, 32);
    dim3 blockPerGrid((width + threadsPerBlock.x - 1)/threadsPerBlock.x, (height + threadsPerBlock.y - 1)/threadsPerBlock.y);

    hipMemcpy(in_gpu, gaussImg.data, memsize, hipMemcpyHostToDevice);

    sobel_gpu<<<blockPerGrid, threadsPerBlock>>>(in_gpu, out_gpu, height, width);

    hipMemcpy(dst_gpu.data, out_gpu, memsize, hipMemcpyDeviceToHost);

    imwrite("save.png", dst_gpu);
    hipFree(in_gpu);
    hipFree(out_gpu);

    return 0;
}