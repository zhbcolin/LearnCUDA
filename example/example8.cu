#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <math.h>
//
//#define N 100000000
//#define BLOCK_SIZE 256
//#define GRID_SIZE 32
//
//__managed__ int source[N];
//__managed__ int gpu_result[1] = {0};
//
//// source[N]: 1 + 2 + 3 + 4 + ... + N
//// cpu: for loop
//// gpu = 1 + 2 + 3 + 4 + ... + N    0 + 1 + 2 + 3 + 4 + 5 + 6 + 7 8 9 10 11 12 ... 31
//// thread id step 0: tid0: source[0] + source[4] -> source[0]
////                   tid1: source[1] + source[5] -> source[1]
////                   tid2: source[2] + source[6] -> source[2]
////                   tid4: source[4] + source[7] -> source[3]
////           step 1: tid0: source[0] + source[2] -> source[0]
////                   tid1: source[1] + source[3] -> source[1]
////
////           step 2: tid0: source[0] + source[1] -> source[0]
//// thread id: blockDim.x * blockIdx.x + threadIdx.x + step * blockDim.x * GridDim.x
//// thread 0: source[0, 8, 16, 24] sum -> shared memory
//
//__global__ void sum_gpu(int *in, int count, int *out) {
//    __shared__ int ken[BLOCK_SIZE];
//    // grid loop
//    int shared_tmp = 0;
//    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x) {
//        shared_tmp += in[idx];
//    }
//    ken[threadIdx.x] = shared_tmp;
//    __syncthreads();
//
//    int tmp = 0;
//    for(int total_threads = BLOCK_SIZE/2; total_threads >= 1; total_threads /= 2) {
//        if(threadIdx.x < total_threads) {
//            tmp = ken[threadIdx.x] + ken[threadIdx.x + total_threads];
//        }
//        __syncthreads();
//        if(threadIdx.x < total_threads) {
//            ken[threadIdx.x] = tmp;
//        }
//    }
//    // block_sum -> share memory[0]
//    if(blockIdx.x * blockDim.x < count) {
//        if(threadIdx.x == 0) {
//            atomicAdd(out, ken[0]);
//            // memory space wmr
//        }
//    }
//}
//
//int main() {
//    int cpu_result = 0;
//
//    printf("Init input source[N]\n");
//    for(int i = 0; i < N; ++i) {
//        source[i] = rand() % 10;
//    }
//
//    hipEvent_t start, stop_cpu, stop_gpu;
//    hipEventCreate(&start);
//    hipEventCreate(&stop_cpu);
//    hipEventCreate(&stop_gpu);
//
//    hipEventRecord(start);
//    hipEventSynchronize(start);
//
//    for(int i = 0; i < 20; ++i) {
//        gpu_result[0] = 0;
//        sum_gpu<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_result);
//        hipDeviceSynchronize();
//    }
//    hipEventRecord(stop_gpu);
//    hipEventSynchronize(stop_gpu);
//
//    for(int i = 0; i < N; ++i) {
//        cpu_result += source[i];
//    }
//
//    hipEventRecord(stop_cpu);
//    hipEventSynchronize(stop_cpu);
//
//    float time_cpu, time_gpu;
//    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);
//    hipEventElapsedTime(&time_gpu, start, stop_gpu);
//
//    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20.0);
//    printf("Result: %s\nGPU_result: %d;\nCPU_result: %d;\n", (gpu_result[0] == cpu_result) ? "Pass" : "Error", gpu_result[0], cpu_result);
//
//    return 0;
//}
#include <stdio.h>
#include <math.h>

#define N 100000000
#define BLOCK_SIZE 256
#define GRID_SIZE 32

__managed__ int source[N];
__managed__ int gpu_result[1] = {0};

// source[N]:  1 + 2 + 3 + 4 + ...............N
// cpu: for loop
// gpu: 1 + 2 + 3 + 4 + ...............N    0 + 1 + 2 + 3 + 4 + 5 + 6 + 7 8 9 10 11 12  .... 31
// thread id step 0:  tid0:source[0] + source[4] -> source[0]
//                    tid1:source[1] + source[5] -> source[1]
//                    tid2:source[2] + source[6] -> source[2]
//                    tid4:source[4] + source[7] -> source[3]
//           step 1:  tid0: source[0] + source[2] -> source[0]
//                    tid1: source[1] + source[3] -> source[1]
//
//           step 2:  tid0: source[0] + source[1] -> source[0]
// thread id: blockDim.x * blockIdx.x + threadIdx.x + step * blockDim.x * GridDim.x
// thread 0: source[0, 8, 16, 24] sum -> shared memory
//
//

__global__ void sum_gpu(int *in, int count, int *out)
{
    __shared__ int ken[BLOCK_SIZE];
    //grid_loop
    int shared_tmp=0;
    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x)
    {
        shared_tmp +=in[idx];
    }
    ken[threadIdx.x] = shared_tmp;
    __syncthreads();

    int tmp =0;
    for(int total_threads = BLOCK_SIZE/2; total_threads>=1; total_threads/=2)
    {
        if(threadIdx.x < total_threads)
        {
            tmp = ken[threadIdx.x] + ken[threadIdx.x + total_threads];
        }
        __syncthreads();
        if(threadIdx.x < total_threads)
        {
            ken[threadIdx.x] = tmp;
        }
    }
    // block_sum -> share memory[0]
    if(blockIdx.x * blockDim.x < count)
    {
        if(threadIdx.x == 0)
        {
            atomicAdd(out, ken[0]);
            // memory space wmr
        }
    }


}



int main()
{
    int cpu_result =0;


    printf("Init input source[N]\n");
    for(int i =0; i<N; i++)
    {
        source[i] = rand()%10;
    }

    hipEvent_t start, stop_cpu, stop_gpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    for(int i = 0; i<20; i++)
    {
        gpu_result[0] = 0;
        sum_gpu<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    for(int i =0; i<N; i++)
    {
        cpu_result +=source[i];
    }

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);
    hipEventElapsedTime(&time_gpu, start, stop_gpu);

    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20);
    printf("Result: %s\nGPU_result: %d;\nCPU_result: %d;\n", (gpu_result[0] == cpu_result)?"Pass":"Error", gpu_result[0], cpu_result);

    return 0;
}`